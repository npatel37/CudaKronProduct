#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include "CrsMatrix.h"
#include <cassert>
#include <stdexcept>
#include <vector>
#include <complex>
#include <cmath> 

typedef float VType;

__global__
void kronk_kernel(int* A_Col,int* B_Col,VType* A_Val,VType* B_Val,int* C_Col,VType* C_Val, int na, int counter, int size_kk, int start_k, int start_kk ) {  

  //** Global Mem is slower than local Mem, so local variables are created so Global Var on accessed once
 int s_size_kk ;
 int s_start_kk ;
 int s_start_k ;
 int s_counter;
 int s_na ;

 s_size_kk = size_kk;
 s_start_kk = start_kk;
 s_start_k = start_k;
 s_counter = counter;
 s_na = na;


    int inx = threadIdx.x + blockDim.x * blockIdx.x;
    int iny = threadIdx.y + blockDim.y * blockIdx.y;



     C_Col[inx + iny*s_size_kk + s_counter] = A_Col[iny + s_start_k] + B_Col[inx + s_start_kk] * s_na;
     C_Val[inx + iny*s_size_kk + s_counter] = A_Val[iny + s_start_k] * B_Val[inx + s_start_kk] ;

 }


void CRS_EXTRN_PROD( CrsMatrix<VType>  &C,CrsMatrix<VType> const &A,CrsMatrix<VType> const &B,VType **pd,int **pi) {

   assert(A.row()==A.col());
   assert(B.row()==B.col());
   int n=A.row()*B.row();
   C.resize(n,n);
   int na = A.row();
   int coa = A.nonZero();
   int cob = B.nonZero();
   int i,alpha,beta,counter=0;
   C.resizecv(coa*cob);

   int *d_A_Col,*d_B_Col,*d_C_Col;
   VType *d_A_Val,*d_B_Val,*d_C_Val;

 // Allocating the arrays on the Device
    hipMalloc((void **) &d_A_Col, sizeof(int) * coa ); 
    hipMalloc((void **) &d_A_Val, sizeof(VType) * coa );  
    hipMalloc((void **) &d_B_Col, sizeof(int) * cob ); 
    hipMalloc((void **) &d_B_Val, sizeof(VType) * cob );  
    hipMalloc((void **) &d_C_Col, sizeof(int) * coa*cob ); 
    hipMalloc((void **) &d_C_Val, sizeof(VType) * coa*cob );  

 // Copying the necessary arrays from the Host to the Device
    hipMemcpy( d_A_Col, &A.colind_[0], sizeof(int) * coa, hipMemcpyHostToDevice );
    hipMemcpy( d_A_Val, &A.values_[0], sizeof(VType) * coa, hipMemcpyHostToDevice );
    hipMemcpy( d_B_Col, &B.colind_[0], sizeof(int) * cob, hipMemcpyHostToDevice );
    hipMemcpy( d_B_Val, &B.values_[0], sizeof(VType) * cob, hipMemcpyHostToDevice );

  
  for (i=0;i<n;i++) {
   C.setRow(i,counter);
   beta = int(i/na);
   alpha = i - beta * na;

   int size_k = A.getRowPtr(alpha+1) - A.getRowPtr(alpha); 
   int size_kk = B.getRowPtr(beta+1) - B.getRowPtr(beta); 
   int start_k = A.getRowPtr(alpha);
   int start_kk = B.getRowPtr(beta);

   //*** Creating the grid and block geometry and launching the Kernel on the Device
   dim3 grid_dim(  1 ,  1);
   dim3 block_dim(size_kk  , size_k);
   kronk_kernel<<<grid_dim, block_dim>>>(d_A_Col, d_B_Col, d_A_Val, d_B_Val, d_C_Col, d_C_Val, na, counter,
                                              size_kk, start_k, start_kk);
   counter = counter + size_kk*size_k;
  }

  //** Copying the Arrays for CrsMatrix "C" from the Device to the Host
   hipMemcpy( &C.colind_[0], d_C_Col, sizeof(int) * coa*cob, hipMemcpyDeviceToHost );
   hipMemcpy( &C.values_[0], d_C_Val, sizeof(VType) * coa*cob, hipMemcpyDeviceToHost );
   C.setRow(n,counter);

   pi[0] = d_B_Col;  // 
   pd[0] = d_B_Val;  // Saves these pointers to devie memory so that
   pi[1] = d_C_Col; // they can be used on the next iterations of CRS_EXTRN_PROD_PARTIAL
   pd[1] = d_C_Val; // 

  hipFree(d_A_Col); // Freeing the Device memory for
  hipFree(d_A_Val); // the CrsMatrix "A" because its no longer needed

}



void CRS_EXTRN_PROD_PARTIAL( CrsMatrix<VType>  &C,CrsMatrix<VType> const &A,CrsMatrix<VType> const &B,VType **pd,int **pi) {

   assert(A.row()==A.col());
   assert(B.row()==B.col());
   int n=A.row()*B.row();
   C.resize(n,n);
   int na = A.row();
   int coa = A.nonZero();
   int cob = B.nonZero();
   int i,alpha,beta,counter=0;
   C.resizecv(coa*cob);

   int *d_A_Col,*d_B_Col,*d_C_Col;
   VType *d_A_Val,*d_B_Val,*d_C_Val;

   d_B_Col= pi[0];
   d_B_Val= pd[0];
   d_A_Col= pi[1];
   d_A_Val= pd[1];

    hipMalloc((void **) &d_C_Col, sizeof(int) * coa*cob ); 
    hipMalloc((void **) &d_C_Val, sizeof(VType) * coa*cob );  

  
  for (i=0;i<n;i++) {
    C.setRow(i,counter);
    beta = int(i/na);
    alpha = i - beta * na;

    int size_k = A.getRowPtr(alpha+1) - A.getRowPtr(alpha); 
    int size_kk = B.getRowPtr(beta+1) - B.getRowPtr(beta); 
    int start_k = A.getRowPtr(alpha);
    int start_kk = B.getRowPtr(beta);

   //*** Creating the grid and block geometry and launching the Kernel on the Device
    dim3 grid_dim(  1 ,  1);
    dim3 block_dim( size_kk , size_k);
    kronk_kernel<<<grid_dim, block_dim>>>(d_A_Col, d_B_Col, d_A_Val, d_B_Val, d_C_Col, d_C_Val, na, counter,
                                            size_kk, start_k, start_kk);
    counter = counter + size_kk*size_k;
  }

  //** Copying the Arrays for CrsMatrix "C" from the Device to the Host
   hipMemcpy( &C.colind_[0], d_C_Col, sizeof(int) * coa*cob, hipMemcpyDeviceToHost );
   hipMemcpy( &C.values_[0], d_C_Val, sizeof(VType) * coa*cob, hipMemcpyDeviceToHost );
   C.setRow(n,counter);

   pi[1] = d_C_Col; // Saves these pointers to devie memory so that
   pd[1] = d_C_Val;  // they can be used on the next iterations of CRS_EXTRN_PROD_PARTIAL

   hipFree(d_A_Col); // Freeing the Device memory for
   hipFree(d_A_Val); // the CrsMatrix "A" because its no longer neede


}


void Free_GPU_MEM(VType **pd,int **pi) {

 //** Freeing the Device memory  **//
     hipFree(pi[0]); 
     hipFree(pi[1]);
     hipFree(pd[0]);
     hipFree(pd[1]);
}























